
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define NUM_THREADS 128  
#define NUM_BLOCKS 8  

__global__
void saxpy(int n, float a, int * x, int * y, int * out)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) out[i] = a*x[i] + y[i];
}


void fill_array(int *data, int N){
    for(int idx=0;idx<N;idx++){
        data[idx] = idx;
    }
}

void print_output(int *x, int *y, int *out, int N){
    for(int idx=0;idx<N;idx++){
        printf("2 X %d + %d = %d\n", x[idx], y[idx], out[idx]);
    }
}

int main(){
    int N = NUM_BLOCKS * NUM_THREADS;
    int size = N * sizeof(int);
    int *x, *y, *out, *d_x, *d_y, *d_out;

    x = (int *)malloc(size); fill_array(x,N);
    y = (int *)malloc(size); fill_array(y,N);
    out = (int *)malloc(size);
    
    hipMalloc(&d_x, size); 
    hipMalloc(&d_y, size); 
    hipMalloc(&d_out, size); 

    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);
    hipMemcpy(d_out, out, size, hipMemcpyHostToDevice);

    // Perform SAXPY on 1M elements
    saxpy<<<NUM_BLOCKS,NUM_THREADS>>>(N, 2.0, d_x, d_y, d_out);

    hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);
    print_output(x, y, out,N);
    free(x); free(y); free(out);
    return 0;
}
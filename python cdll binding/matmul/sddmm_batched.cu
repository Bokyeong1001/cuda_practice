
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h> // cudaMalloc, cudaMemcpy, etc.
#include <hipsparse.h>         // cusparseSpMM
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

//nvcc -Xcompiler -fPIC -shared -lcusparse -o sddmm_batched.so sddmm_batched.cu

extern "C" {
void sddmm_batched(float *hA, float *hB, float *hC_values, int *hC_offsets, int *hC_columns, int m, int n, int k, int C_nnz, int num_batches)
{
    // Host problem definition
    int   A_num_rows   = m;
    int   A_num_cols   = k;
    int   B_num_rows   = A_num_cols;
    int   B_num_cols   = n;
    int   lda          = A_num_cols;
    int   ldb          = B_num_cols;
    int   A_size       = lda * A_num_rows;
    int   B_size       = ldb * B_num_rows;

    float alpha        = 1.0f;
    float beta         = 0.0f;

    int   *dC_offsets, *dC_columns;
    float *dC_values, *dB, *dA;

    hipMalloc((void**) &dA, A_size * num_batches * sizeof(float));
    hipMalloc((void**) &dB, B_size * num_batches * sizeof(float));
    hipMalloc((void**) &dC_offsets, (A_num_rows + 1) * sizeof(int));
    hipMalloc((void**) &dC_columns, C_nnz * num_batches * sizeof(int));
    hipMalloc((void**) &dC_values, C_nnz * num_batches * sizeof(float));

    hipMemcpy(dA, hA, A_size * sizeof(float) * num_batches, hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, B_size * sizeof(float) * num_batches, hipMemcpyHostToDevice);
    hipMemcpy(dC_offsets, hC_offsets, (A_num_rows + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dC_columns, hC_columns, C_nnz * sizeof(int) * num_batches, hipMemcpyHostToDevice);
    hipMemcpy(dC_values, hC_values, C_nnz * sizeof(float) * num_batches, hipMemcpyHostToDevice);

    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseDnMatDescr_t matA, matB;
    hipsparseSpMatDescr_t matC;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    hipsparseCreate(&handle);
    // Create dense matrix A
    hipsparseCreateDnMat(&matA, A_num_rows, A_num_cols, lda, dA,
                                        HIP_R_32F, HIPSPARSE_ORDER_ROW);
    hipsparseDnMatSetStridedBatch(matA, num_batches, A_size);
    // Create dense matrix B
    hipsparseCreateDnMat(&matB, A_num_cols, B_num_cols, ldb, dB,
                                        HIP_R_32F, HIPSPARSE_ORDER_ROW);
    hipsparseDnMatSetStridedBatch(matB, num_batches, B_size);
    // Create sparse matrix C in CSR format
    hipsparseCreateCsr(&matC, A_num_rows, B_num_cols, C_nnz,
                                      dC_offsets, dC_columns, dC_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
    hipsparseCsrSetStridedBatch(matC, num_batches, 0, C_nnz);
    // allocate an external buffer if needed
    hipsparseSDDMM_bufferSize(
                                 handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SDDMM_ALG_DEFAULT, &bufferSize);
    hipMalloc(&dBuffer, bufferSize);

    // execute preprocess (optional)
    hipsparseSDDMM_preprocess(
                                  handle,
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                  HIPSPARSE_SDDMM_ALG_DEFAULT, dBuffer);
    // execute SpMM
    hipsparseSDDMM(handle,
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                  HIPSPARSE_SDDMM_ALG_DEFAULT, dBuffer);
    // destroy matrix/vector descriptors
    hipsparseDestroyDnMat(matA);
    hipsparseDestroyDnMat(matB);
    hipsparseDestroySpMat(matC);
    hipsparseDestroy(handle);

    hipMemcpy(hC_values, dC_values, C_nnz * sizeof(float) * num_batches,
                           hipMemcpyDeviceToHost);

    //--------------------------------------------------------------------------
    // device memory deallocation
    hipFree(dBuffer);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC_offsets);
    hipFree(dC_columns);
    hipFree(dC_values);
}
}

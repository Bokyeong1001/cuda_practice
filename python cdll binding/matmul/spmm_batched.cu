
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h> // cudaMalloc, cudaMemcpy, etc.
#include <hipsparse.h>         // cusparseSpMM
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <math.h>             // fabs

//nvcc -Xcompiler -fPIC -shared -lcusparse -o spmm_batched.so spmm_batched.cu

extern "C" {
void spmm_batched(float *hA_values, float *hB, float *hC, int *hA_csrOffsets, int *hA_columns, int m, int n, int k, int A_nnz, int num_batches)
{
    // Host problem definition
    int   A_num_rows  = m;
    int   A_num_cols  = k;
    int   B_num_rows  = k;
    int   B_num_cols  = n;
    int   ldb         = B_num_rows;
    int   ldc         = A_num_rows;
    int   B_size      = ldb * B_num_cols;
    int   C_size      = ldc * B_num_cols;

    float alpha            = 1.0f;
    float beta             = 0.0f;

    //--------------------------------------------------------------------------
    // Device memory management
    int   *dA_csrOffsets, *dA_columns;
    float *dA_values, *dB, *dC;
    hipMalloc((void**) &dA_csrOffsets,
                           (A_num_rows + 1) * sizeof(int));
    hipMalloc((void**) &dA_columns,
                           A_nnz * num_batches * sizeof(int));
    hipMalloc((void**) &dA_values,
                           A_nnz * num_batches * sizeof(float));
    hipMalloc((void**) &dB,
                           B_size * num_batches * sizeof(float));
    hipMalloc((void**) &dC,
                           C_size * num_batches * sizeof(float));

    hipMemcpy(dA_csrOffsets, hA_csrOffsets,
                           (A_num_rows + 1) * sizeof(int),
                           hipMemcpyHostToDevice);
    hipMemcpy(dA_columns, hA_columns, A_nnz * num_batches * sizeof(int),
                           hipMemcpyHostToDevice);
    hipMemcpy(dA_values, hA_values, A_nnz * num_batches * sizeof(float),
                           hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, B_size * num_batches * sizeof(float),
                           hipMemcpyHostToDevice);
    hipMemcpy(dC, hC, C_size * num_batches * sizeof(float),
                           hipMemcpyHostToDevice);
    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB, matC;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    hipsparseCreate(&handle);
    // Create sparse matrix A in CSR format
    hipsparseCreateCsr(&matA, A_num_rows, A_num_cols, A_nnz,
                                      dA_csrOffsets, dA_columns, dA_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
    hipsparseCsrSetStridedBatch(matA, num_batches, 0, A_nnz);
    // Alternatively, the following code can be used for matA broadcast
    // cusparseCsrSetStridedBatch(matA, num_batches, 0, 0);
    // Create dense matrix B
    hipsparseCreateDnMat(&matB, A_num_cols, B_num_cols, ldb, dB,
                                        HIP_R_32F, HIPSPARSE_ORDER_COL);
    hipsparseDnMatSetStridedBatch(matB, num_batches, B_size);
    // Create dense matrix C
    hipsparseCreateDnMat(&matC, A_num_rows, B_num_cols, ldc, dC,
                                        HIP_R_32F, HIPSPARSE_ORDER_COL);
    hipsparseDnMatSetStridedBatch(matC, num_batches, C_size);

    // allocate an external buffer if needed
    hipsparseSpMM_bufferSize(
                                 handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SPMM_CSR_ALG2, &bufferSize);
    hipMalloc(&dBuffer, bufferSize);

    // execute SpMM
    hipsparseSpMM(handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SPMM_CSR_ALG2, dBuffer);

    // destroy matrix/vector descriptors
    hipsparseDestroySpMat(matA);
    hipsparseDestroyDnMat(matB);
    hipsparseDestroyDnMat(matC);
    hipsparseDestroy(handle);
    //--------------------------------------------------------------------------
    // device result check
    hipMemcpy(hC, dC, C_size * num_batches * sizeof(float),
                           hipMemcpyDeviceToHost);
    //--------------------------------------------------------------------------
    // device memory deallocation
    hipFree(dBuffer);
    hipFree(dA_csrOffsets);
    hipFree(dA_columns);
    hipFree(dA_values);
    hipFree(dB);
    hipFree(dC);
}
}
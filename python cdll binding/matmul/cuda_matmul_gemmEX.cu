#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipblas.h>
#include <hipblas.h>

//nvcc -Xcompiler -fPIC -shared -lcublas -o cuda_matmul_gemmEX.so cuda_matmul_gemmEX.cu

void gpu_blas_mmul(const float *A, const float *B, float *C, const int m, const int n, const int k) {
    int lda=m,ldb=k,ldc=m;
    const float alf = 1;
    const float bet = 0;
    const float *alpha = &alf;
    const float *beta = &bet;
 
    // Create a handle for CUBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);
 
    // Do the actual multiplication
    hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, HIP_R_32F, lda, B, HIP_R_32F, ldb, beta, C, HIP_R_32F, ldc, HIPBLAS_COMPUTE_32F_FAST_16F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
 
    // Destroy the handle
    hipblasDestroy(handle);
}

extern "C" {
void cuda_matmul(float *a, float *b, float *c, int m, int n, int k, size_t input_size, size_t weight_size, size_t output_size)
{
    float *d_a, *d_b, *d_c;

    hipMalloc((void **)&d_a, input_size * sizeof(float));
    hipMalloc((void **)&d_b, weight_size * sizeof(float));
    hipMalloc((void **)&d_c, output_size * sizeof(float));

    hipMemcpy(d_a, a, input_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, weight_size * sizeof(float), hipMemcpyHostToDevice);

    gpu_blas_mmul(d_a, d_b, d_c, m, n, k);

    hipMemcpy(c, d_c, output_size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
}

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h> // cudaMalloc, cudaMemcpy, etc.
#include <hipsparse.h>         // cusparseSpMM
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

//nvcc -Xcompiler -fPIC -shared -lcusparse -o attn_forward.so attn_forward.cu

__global__ void scale_softmax_kernel(float *d_out, float *d_values, int rows, int cols, int emb_dim, int block_size, int num_batch)
{
    float scale = sqrtf(float(emb_dim));
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= rows*num_batch)
        return;

    for (int k = 0; k < block_size; k++){
            d_values[i * block_size + k] = d_values[i * block_size + k]/scale;
    }

    float max = 0.0;
    for (int k = 0; k < block_size; k++){
        if (max < d_values[i * block_size + k]){
            max = d_values[i * block_size + k];
        }
    }
    float sum = 0.0f;
    for (int k = 0; k < cols; k++){
        if (k < block_size){
            sum += expf(d_values[i * block_size + k] - max);
        }
        else{
            sum += expf(0 - max);
        }
    }
    for (int k = 0; k < block_size; k++){
        d_out[i * block_size + k] = expf(d_values[i * block_size + k] - max) / sum;
    }
}

void spmm(hipsparseHandle_t handle, hipsparseOperation_t opA, void *dBuffer, float *dA, float *dB, float *dC, int *d_offsets, int *d_columns, int seq_len, int emb_dim, int nnz, int num_batches)
{
    // Host problem definition
    int   ldb         = seq_len;
    int   ldc         = seq_len;
    int   output_size = seq_len * emb_dim;

    float alpha            = 1.0f;
    float beta             = 0.0f;

    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB, matC;
    size_t               bufferSize = 0;

    // Create sparse matrix A in CSR format
    hipsparseCreateCsr(&matA, seq_len, seq_len, nnz,
                                      d_offsets, d_columns, dA,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
    hipsparseCsrSetStridedBatch(matA, num_batches, 0, nnz);
    // Alternatively, the following code can be used for matA broadcast
    // cusparseCsrSetStridedBatch(matA, num_batches, 0, 0);
    // Create dense matrix B
    hipsparseCreateDnMat(&matB, seq_len, emb_dim, ldb, dB,
                                        HIP_R_32F, HIPSPARSE_ORDER_COL);
    hipsparseDnMatSetStridedBatch(matB, num_batches, output_size);
    // Create dense matrix C
    hipsparseCreateDnMat(&matC, seq_len, emb_dim, ldc, dC,
                                        HIP_R_32F, HIPSPARSE_ORDER_COL);
    hipsparseDnMatSetStridedBatch(matC, num_batches, output_size);

    // allocate an external buffer if needed
    hipsparseSpMM_bufferSize(handle,
                            opA, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                            &alpha, matA, matB, &beta, matC, HIP_R_32F,
                            HIPSPARSE_SPMM_CSR_ALG2, &bufferSize);
    hipMalloc(&dBuffer, bufferSize);

    // execute SpMM
    hipsparseSpMM(handle, 
                opA, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                &alpha, matA, matB, &beta, matC, HIP_R_32F,
                HIPSPARSE_SPMM_CSR_ALG2, dBuffer);

    // destroy matrix/vector descriptors
    hipsparseDestroySpMat(matA);
    hipsparseDestroyDnMat(matB);
    hipsparseDestroyDnMat(matC);
    hipFree(dBuffer);
}

void sddmm(hipsparseHandle_t handle, void *dBuffer, float *dQuery, float *dKey, float *dAttn, int *d_offsets, int *d_columns, int seq_len, int emb_dim, int nnz, int num_batches){
    
    size_t bufferSize  = 0;
    int   lda          = emb_dim;
    int   ldb          = seq_len;
    int   input_size       = seq_len * emb_dim;

    float alpha            = 1.0f;
    float beta             = 0.0f;

    hipsparseDnMatDescr_t matA, matB;
    hipsparseSpMatDescr_t matC;
    
    // Create dense matrix A
    hipsparseCreateDnMat(&matA, seq_len, emb_dim, lda, dQuery,
                                        HIP_R_32F, HIPSPARSE_ORDER_ROW);
    hipsparseDnMatSetStridedBatch(matA, num_batches, input_size);
    // Create dense matrix B
    hipsparseCreateDnMat(&matB, emb_dim, seq_len, ldb, dKey,
                                        HIP_R_32F, HIPSPARSE_ORDER_ROW);
    hipsparseDnMatSetStridedBatch(matB, num_batches, input_size);
    // Create sparse matrix C in CSR format
    hipsparseCreateCsr(&matC, seq_len, seq_len, nnz,
                                      d_offsets, d_columns, dAttn,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
    hipsparseCsrSetStridedBatch(matC, num_batches, 0, nnz);
    // allocate an external buffer if needed
    hipsparseSDDMM_bufferSize(
                                 handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SDDMM_ALG_DEFAULT, &bufferSize);
    hipMalloc(&dBuffer, bufferSize);

    // execute preprocess (optional)
    hipsparseSDDMM_preprocess(
                                  handle,
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                  HIPSPARSE_SDDMM_ALG_DEFAULT, dBuffer);
    // execute SpMM
    hipsparseSDDMM(handle,
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                  HIPSPARSE_SDDMM_ALG_DEFAULT, dBuffer);
    // destroy matrix/vector descriptors
    hipsparseDestroyDnMat(matA);
    hipsparseDestroyDnMat(matB);
    hipsparseDestroySpMat(matC);
    hipFree(dBuffer);
}

extern "C" {
void attn_forward(float *hQuery, float *hKey, float *hValue, float *hAttn, float *hAttnScore, float *hOut, int *hOffsets, int *hColumns, int seq_len, int emb_dim, int nnz, int block_size, int num_batches)
{
    // Host problem definition
    int   input_size       = seq_len * emb_dim;

    int   *dOffsets, *dColumns;
    float *dQuery, *dKey, *dValue, *dAttn, *dAttnScore, *dOut;

    hipMalloc((void**) &dQuery, input_size * num_batches * sizeof(float));
    hipMalloc((void**) &dKey, input_size * num_batches * sizeof(float));
    hipMalloc((void**) &dValue, input_size * num_batches * sizeof(float));
    hipMalloc((void**) &dAttn, nnz * num_batches * sizeof(float));
    hipMalloc((void**) &dAttnScore, nnz * num_batches * sizeof(float));
    hipMalloc((void**) &dOffsets, (seq_len + 1) * sizeof(int));
    hipMalloc((void**) &dColumns, nnz * num_batches * sizeof(int));
    hipMalloc((void**) &dOut, input_size * num_batches * sizeof(float));

    hipMemcpy(dQuery, hQuery, input_size * sizeof(float) * num_batches, hipMemcpyHostToDevice);
    hipMemcpy(dKey, hKey, input_size * sizeof(float) * num_batches, hipMemcpyHostToDevice);
    hipMemcpy(dValue, hValue, input_size * sizeof(float) * num_batches, hipMemcpyHostToDevice);
    hipMemcpy(dAttn, hAttn, nnz * sizeof(float) * num_batches, hipMemcpyHostToDevice);
    hipMemcpy(dAttnScore, hAttnScore, nnz * sizeof(float) * num_batches, hipMemcpyHostToDevice);
    hipMemcpy(dOffsets, hOffsets, (seq_len + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dColumns, hColumns, nnz * sizeof(int) * num_batches, hipMemcpyHostToDevice);
    hipMemcpy(dOut, hOut, input_size * sizeof(float) * num_batches, hipMemcpyHostToDevice);

    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseCreate(&handle);
    void* dBuffer    = NULL;

    sddmm(handle, dBuffer, dQuery, dKey, dAttn, dOffsets, dColumns, seq_len, emb_dim, nnz, num_batches);
    hipMemcpy(hAttn, dAttn, nnz * sizeof(float) * num_batches, hipMemcpyDeviceToHost);

    scale_softmax_kernel<<<num_batches, seq_len>>>(dAttnScore, dAttn, seq_len, seq_len, emb_dim, block_size, num_batches);
    hipMemcpy(hAttnScore, dAttnScore, nnz * sizeof(float) * num_batches, hipMemcpyDeviceToHost);

    spmm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, dBuffer, dAttnScore, dValue, dOut, dOffsets, dColumns, seq_len, emb_dim, nnz, num_batches);
    hipMemcpy(hOut, dOut, input_size * sizeof(float) * num_batches, hipMemcpyDeviceToHost);
    //--------------------------------------------------------------------------
    // device memory deallocation
    
    hipFree(dQuery);
    hipFree(dKey);
    hipFree(dValue);
    hipFree(dAttn);
    hipFree(dAttnScore);
    hipFree(dOut);
    hipFree(dOffsets);
    hipFree(dColumns);
    
    hipsparseDestroy(handle);
}
}

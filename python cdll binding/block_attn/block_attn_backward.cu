
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h> // cudaMalloc, cudaMemcpy, etc.
#include <hipsparse.h>         // cusparseSpMM
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <math.h>

//nvcc -Xcompiler -fPIC -shared -lcusparse -o attn_backward.so attn_backward.cu

__global__ void softmax_scale_backward_kernel(float *dGradAttnScore, float *dAttnScore, float *dGradAttnScoreScale, float *dGradSum, float *dGradAttnScale,
                                            float *dGradAttn, int seq_len, int emb_dim, int block_size, int num_batches)
{
    float scale = sqrtf(float(emb_dim));
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= seq_len*num_batches)
        return;

    for (int k = 0; k < block_size; k++){
        dGradAttnScoreScale[i * block_size + k] = dGradAttnScore[i * block_size + k] * dAttnScore[i * block_size + k];
    }

    for (int k = 0; k < block_size; k++){
        dGradSum[i] += dGradAttnScoreScale[i * block_size + k];

    }
    for (int k = 0; k < block_size; k++){
        dGradAttn[i * block_size + k] = (dGradAttnScoreScale[i * block_size + k] - (dAttnScore[i * block_size + k] * dGradSum[i]))/scale;
    }
}

void sddmm(hipsparseHandle_t handle, void *dBuffer, float *dQuery, float *dKey, float *dAttn, int *d_offsets, int *d_columns, int seq_len, int emb_dim, int nnz, int num_batches){
    
    size_t bufferSize  = 0;
    int   lda          = emb_dim;
    int   ldb          = seq_len;
    int   input_size       = seq_len * emb_dim;

    float alpha            = 1.0f;
    float beta             = 0.0f;

    hipsparseDnMatDescr_t matA, matB;
    hipsparseSpMatDescr_t matC;
    
    // Create dense matrix A
    hipsparseCreateDnMat(&matA, seq_len, emb_dim, lda, dQuery,
                                        HIP_R_32F, HIPSPARSE_ORDER_ROW);
    hipsparseDnMatSetStridedBatch(matA, num_batches, input_size);
    // Create dense matrix B
    hipsparseCreateDnMat(&matB, emb_dim, seq_len, ldb, dKey,
                                        HIP_R_32F, HIPSPARSE_ORDER_ROW);
    hipsparseDnMatSetStridedBatch(matB, num_batches, input_size);
    // Create sparse matrix C in CSR format
    hipsparseCreateCsr(&matC, seq_len, seq_len, nnz,
                                      d_offsets, d_columns, dAttn,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
    hipsparseCsrSetStridedBatch(matC, num_batches, 0, nnz);
    // allocate an external buffer if needed
    hipsparseSDDMM_bufferSize(
                                 handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SDDMM_ALG_DEFAULT, &bufferSize);
    hipMalloc(&dBuffer, bufferSize);

    // execute preprocess (optional)
    hipsparseSDDMM_preprocess(
                                  handle,
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                  HIPSPARSE_SDDMM_ALG_DEFAULT, dBuffer);
    // execute SpMM
    hipsparseSDDMM(handle,
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                  HIPSPARSE_SDDMM_ALG_DEFAULT, dBuffer);
    // destroy matrix/vector descriptors
    hipsparseDestroyDnMat(matA);
    hipsparseDestroyDnMat(matB);
    hipsparseDestroySpMat(matC);
    hipFree(dBuffer);
}

void spmm(hipsparseHandle_t handle, hipsparseOperation_t opA, void *dBuffer, float *dAttn, float *dValue, float *dOut, int *d_offsets, int *d_columns, int seq_len, int emb_dim, int nnz, int num_batches)
{
    // Host problem definition
    int   ldb         = seq_len;
    int   ldc         = seq_len;
    int   output_size = seq_len * emb_dim;

    float alpha            = 1.0f;
    float beta             = 0.0f;

    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB, matC;
    size_t               bufferSize = 0;

    // Create sparse matrix A in CSR format
    hipsparseCreateCsr(&matA, seq_len, seq_len, nnz,
                                      d_offsets, d_columns, dAttn,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
    hipsparseCsrSetStridedBatch(matA, num_batches, 0, nnz);
    // Alternatively, the following code can be used for matA broadcast
    // cusparseCsrSetStridedBatch(matA, num_batches, 0, 0);
    // Create dense matrix B
    hipsparseCreateDnMat(&matB, seq_len, emb_dim, ldb, dValue,
                                        HIP_R_32F, HIPSPARSE_ORDER_COL);
    hipsparseDnMatSetStridedBatch(matB, num_batches, output_size);
    // Create dense matrix C
    hipsparseCreateDnMat(&matC, seq_len, emb_dim, ldc, dOut,
                                        HIP_R_32F, HIPSPARSE_ORDER_COL);
    hipsparseDnMatSetStridedBatch(matC, num_batches, output_size);

    // allocate an external buffer if needed
    hipsparseSpMM_bufferSize(handle,
                            opA, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                            &alpha, matA, matB, &beta, matC, HIP_R_32F,
                            HIPSPARSE_SPMM_CSR_ALG2, &bufferSize);
    hipMalloc(&dBuffer, bufferSize);

    // execute SpMM
    hipsparseSpMM(handle, 
                opA, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                &alpha, matA, matB, &beta, matC, HIP_R_32F,
                HIPSPARSE_SPMM_CSR_ALG2, dBuffer);

    // destroy matrix/vector descriptors
    hipsparseDestroySpMat(matA);
    hipsparseDestroyDnMat(matB);
    hipsparseDestroyDnMat(matC);
    hipFree(dBuffer);
}

extern "C" {
void attn_backward(float *hQuery, float *hKey, float *hValue, float *hAttnScore, float *hGradOutput, float *hGradAttnScore, float *hGradAttnScoreScale, 
                    float *hGradSum, float *hGradAttnScale, float *hGradAttn, float *hGradQuery, float *hGradKey, float *hGradValue,
                    int *hOffsets, int *hColumns, int seq_len, int emb_dim, int nnz, int block_size, int num_batches)
{
    int input_size = seq_len * emb_dim;

    int   *dOffsets, *dColumns;
    float *dQuery, *dKey, *dValue, *dAttnScore, *dGradOutput, *dGradAttnScore, *dGradAttnScoreScale, *dGradSum, *dGradAttnScale, *dGradAttn, *dGradQuery, *dGradKey, *dGradValue;

    hipMalloc((void**) &dQuery, input_size * num_batches * sizeof(float));
    hipMalloc((void**) &dKey, input_size * num_batches * sizeof(float));
    hipMalloc((void**) &dValue, input_size * num_batches * sizeof(float));
    hipMalloc((void**) &dAttnScore, nnz * num_batches * sizeof(float));
    hipMalloc((void**) &dGradOutput, input_size * num_batches * sizeof(float));
    hipMalloc((void**) &dGradAttnScore, nnz * num_batches * sizeof(float));
    hipMalloc((void**) &dGradAttnScoreScale, nnz * num_batches * sizeof(float));
    hipMalloc((void**) &dGradSum, seq_len * num_batches * sizeof(float));
    hipMalloc((void**) &dGradAttnScale, nnz * num_batches * sizeof(float));
    hipMalloc((void**) &dGradAttn, nnz * num_batches * sizeof(float));
    hipMalloc((void**) &dGradQuery, input_size * num_batches * sizeof(float));
    hipMalloc((void**) &dGradKey, input_size * num_batches * sizeof(float));
    hipMalloc((void**) &dGradValue, input_size * num_batches * sizeof(float));

    hipMalloc((void**) &dOffsets, (seq_len + 1) * sizeof(int));
    hipMalloc((void**) &dColumns, nnz * num_batches * sizeof(int));

    hipMemcpy(dQuery, hQuery, input_size * sizeof(float) * num_batches, hipMemcpyHostToDevice);
    hipMemcpy(dKey, hKey, input_size * num_batches * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dValue, hValue, input_size * num_batches * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dAttnScore, hAttnScore, nnz * num_batches * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dGradOutput, hGradOutput, input_size * num_batches * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(dOffsets, hOffsets, (seq_len + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dColumns, hColumns, nnz * sizeof(int) * num_batches, hipMemcpyHostToDevice);

    hipsparseHandle_t     handle = NULL;
    hipsparseCreate(&handle);
    void* dBuffer    = NULL;

    sddmm(handle, dBuffer, dGradOutput, dValue, dGradAttnScore, dOffsets, dColumns, seq_len, emb_dim, nnz, num_batches);
    hipMemcpy(hGradAttnScore, dGradAttnScore, nnz * sizeof(float) * num_batches, hipMemcpyDeviceToHost);

    softmax_scale_backward_kernel<<<num_batches, seq_len>>>(dGradAttnScore, dAttnScore, dGradAttnScoreScale, dGradSum, dGradAttnScale, dGradAttn, seq_len, emb_dim, block_size, num_batches);
    hipMemcpy(hGradAttnScoreScale, dGradAttnScoreScale, nnz * sizeof(float) * num_batches, hipMemcpyDeviceToHost);
    hipMemcpy(hGradSum, dGradSum, seq_len * sizeof(float) * num_batches, hipMemcpyDeviceToHost);
    hipMemcpy(hGradAttn, dGradAttn, nnz * sizeof(float) * num_batches, hipMemcpyDeviceToHost);

    spmm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, dBuffer, hGradAttn, dKey, dGradQuery, dOffsets, dColumns, seq_len, emb_dim, nnz, num_batches);
    hipMemcpy(hGradQuery, dGradQuery, input_size * sizeof(float) * num_batches, hipMemcpyDeviceToHost);
    
    spmm(handle, HIPSPARSE_OPERATION_TRANSPOSE, dBuffer, hGradAttn, dQuery, dGradKey, dOffsets, dColumns, seq_len, emb_dim, nnz, num_batches);
    hipMemcpy(hGradKey, dGradKey, input_size * sizeof(float) * num_batches, hipMemcpyDeviceToHost);

    spmm(handle, HIPSPARSE_OPERATION_TRANSPOSE, dBuffer, dAttnScore, dGradOutput, dGradValue, dOffsets, dColumns, seq_len, emb_dim, nnz, num_batches);
    hipMemcpy(hGradValue, dGradValue, input_size * sizeof(float) * num_batches, hipMemcpyDeviceToHost);

    hipFree(dQuery);
    hipFree(dKey);
    hipFree(dValue);
    hipFree(dAttnScore);
    hipFree(dGradOutput);
    hipFree(dGradAttnScore);
    hipFree(dGradAttnScoreScale);
    hipFree(dGradSum);
    hipFree(dGradAttnScale);
    hipFree(dGradAttn);
    hipFree(dGradQuery);
    hipFree(dGradKey);
    hipFree(dGradValue);
    hipFree(dOffsets);
    hipFree(dColumns);
    
    hipsparseDestroy(handle);
}
}

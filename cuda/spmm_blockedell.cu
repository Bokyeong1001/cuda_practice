
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>        // data types
#include <hip/hip_runtime_api.h> // cudaMalloc, cudaMemcpy, etc.
#include <hipsparse.h>         // cusparseSpMM
#include <cstdio>            // printf
#include <cstdlib>           // EXIT_FAILURE

//nvcc -lcusparse -o spmm_blockedell spmm_blockedell.cu

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        std::printf("CUDA API failed at line %d with error: %s (%d)\n",        \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        std::printf("CUSPARSE API failed at line %d with error: %s (%d)\n",    \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

void print_matrix(const float *A, int nr_rows_A, int nr_cols_A) {
 
    for(int i = 0; i < nr_rows_A; i++){
        for(int j = 0; j < nr_cols_A; j++){
            printf("%0.1f ",A[j * nr_rows_A + i]);
        }
        printf("\n");
    }
    printf("\n");
}

void print_block_matrix(const float *hA_values, const int *hA_columns, int A_num_rows, int A_num_cols, int A_ell_blocksize, int A_ell_cols, int A_num_blocks) {
    int k = 0;
    int tmp = 0;
    bool block = false;
    bool init = true;
    for(int i = 0; i < A_num_rows/A_ell_blocksize; i++){
        init = true;
        for(int r=0;r<A_ell_blocksize; r++){
            if(init){
                k=A_num_blocks/A_ell_cols*i;
            }
            else{
                k=0;
            }
            for(int bc=0;bc<A_ell_cols;bc++){
                if(bc==hA_columns[k]){
                    if(k>A_num_blocks/A_ell_cols*i){
                        for(int br = 0; br < A_ell_blocksize; br++){
                            printf("0.0 ");
                        }   
                        break;
                    }
                    block = true;
                    for(int br = 0; br < A_ell_blocksize; br++){
                        printf("%.01f ", hA_values[tmp]);
                        tmp++;
                    }
                    if(block){
                        k++;
                        block=false;
                    }
                }
                else{
                    for(int br = 0; br < A_ell_blocksize; br++){
                        printf("0.0 ");
                    }
                }
            }
            printf("\n");
        }
    }
    printf("\n");
}

const int EXIT_UNSUPPORTED = 2;

int main() {
    // Host problem definition
    int   A_num_rows      = 4;
    int   A_num_cols      = 4;
    int   A_ell_blocksize = 2;
    int   A_ell_cols      = 2;
    int   A_num_blocks    = A_ell_cols * A_num_rows /
                           (A_ell_blocksize * A_ell_blocksize);
    int   B_num_rows      = A_num_cols;
    int   B_num_cols      = 3;
    int   ldb             = B_num_rows;
    int   ldc             = A_num_rows;
    int   B_size          = ldb * B_num_cols;
    int   C_size          = ldc * B_num_cols;
    int   hA_columns[]    = {1,0};
    float alpha           = 1.0f;
    float beta            = 0.0f;
    //--------------------------------------------------------------------------
    // Check compute capability
    hipDeviceProp_t props;
    CHECK_CUDA( hipGetDeviceProperties(&props, 0) )
    if (props.major < 7) {
      std::printf("hipsparseSpMM with blocked ELL format is supported only "
                  "with compute capability at least 7.0\n");
      return EXIT_UNSUPPORTED;
    }

    float *hA_values, *hB, *hC;
    hipHostMalloc((void **) &hA_values, sizeof(float) * A_ell_blocksize * A_ell_blocksize * A_num_blocks, hipHostMallocDefault);
    hipHostMalloc((void **) &hB, sizeof(float) * B_num_rows * B_num_cols, hipHostMallocDefault);
    hipHostMalloc((void **) &hC, sizeof(float) * A_num_rows * B_num_cols, hipHostMallocDefault);

    for (int i = 0; i < A_ell_blocksize * A_ell_blocksize * A_num_blocks; ++i) {
        hA_values[i] = i+1;
    }

    for (int i = 0; i < B_num_rows; ++i) {
        for (int j = 0; j < B_num_cols; ++j) {
            hB[j * B_num_rows + i] = j * B_num_rows + i + 1;
        }
    }

    printf("A = \n");
    print_block_matrix(hA_values, hA_columns, A_num_rows, A_num_cols, A_ell_blocksize, A_ell_cols, A_num_blocks);

    printf("B = \n");
    print_matrix(hB, B_num_rows, B_num_cols);

    //--------------------------------------------------------------------------
    // Device memory management
    int    *dA_columns;
    float *dA_values, *dB, *dC;
    CHECK_CUDA( hipMalloc((void**) &dA_columns, A_num_blocks * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dA_values,
                                    A_ell_cols * A_num_rows * sizeof(float)) )
    CHECK_CUDA( hipMalloc((void**) &dB, B_size * sizeof(float)) )
    CHECK_CUDA( hipMalloc((void**) &dC, C_size * sizeof(float)) )

    CHECK_CUDA( hipMemcpy(dA_columns, hA_columns,
                           A_num_blocks * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_values, hA_values,
                           A_ell_cols * A_num_rows * sizeof(float),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dB, hB, B_size * sizeof(float),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dC, hC, C_size * sizeof(float),
                           hipMemcpyHostToDevice) )
    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB, matC;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create sparse matrix A in blocked ELL format
    CHECK_CUSPARSE( hipsparseCreateBlockedEll(
                                      &matA,
                                      A_num_rows, A_num_cols, A_ell_blocksize,
                                      A_ell_cols, dA_columns, dA_values,
                                      HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    // Create dense matrix B
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matB, A_num_cols, B_num_cols, ldb, dB,
                                        HIP_R_32F, HIPSPARSE_ORDER_COL) )
    // Create dense matrix C
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matC, A_num_rows, B_num_cols, ldc, dC,
                                        HIP_R_32F, HIPSPARSE_ORDER_COL) )
    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSpMM_bufferSize(
                                 handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize) )
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

    // execute SpMM
    CHECK_CUSPARSE( hipsparseSpMM(handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer) )

    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matB) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matC) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    //--------------------------------------------------------------------------
    // device result check
    CHECK_CUDA( hipMemcpy(hC, dC, C_size * sizeof(float),
                           hipMemcpyDeviceToHost) )
    printf("C = \n");
    print_matrix(hC, A_num_rows, B_num_cols);

    /*int correct = 1;
    for (int i = 0; i < A_num_rows; i++) {
        for (int j = 0; j < B_num_cols; j++) {
            float c_value  = static_cast<float>(hC[i + j * ldc]);
            float c_result = static_cast<float>(hC_result[i + j * ldc]);
            if (c_value != c_result) {
                correct = 0; // direct floating point comparison is not reliable
                break;
            }
        }
    }
    if (correct)
        std::printf("spmm_blockedell_example test PASSED\n");
    else
        std::printf("spmm_blockedell_example test FAILED: wrong result\n");*/
    //--------------------------------------------------------------------------
    // device memory deallocation
    CHECK_CUDA( hipFree(dBuffer) )
    CHECK_CUDA( hipFree(dA_columns) )
    CHECK_CUDA( hipFree(dA_values) )
    CHECK_CUDA( hipFree(dB) )
    CHECK_CUDA( hipFree(dC) )
    return EXIT_SUCCESS;
}
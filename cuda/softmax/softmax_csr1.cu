
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define ROWS 6
#define COLS 6
#define BLOCK_SIZE 2
//nvcc -o softmax_csr1 softmax_csr1.cu

__global__ void softmax(float *d_out, float *d_values, int *d_columns, int *d_csrOffsets, int rows, int cols, int block_size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= rows)
        return;

    float sum = 0.0f;
    for (int k = 0; k < cols; k++){
        if (k < block_size){
            sum += expf(d_values[i * block_size + k]);
        }
        else{
            sum += expf(0);
        }
    }
    
    for (int j = 0; j < cols; j++){
        d_out[i * cols + j] = expf(0) / sum;
    }
    for (int j = 0; j < cols; j++){
        if (j < block_size){
            int idx = d_columns[i * block_size + j];
            d_out[i * cols + idx] = expf(d_values[i * block_size + j]) / sum;
        }
    }
}

int main()
{
    float h_values[ROWS*BLOCK_SIZE] = { 1.0f, 2.0f, 3.0f, 4.0f, 5.0f, 6.0f, 7.0f, 8.0f, 9.0f, 10.0f, 11.0f, 12.0f };
    int   h_csrOffsets[]  = { 0, 2, 4, 6, 8, 10, 12 };
    int   h_columns[]    = { 0, 1, 0, 1, 2, 3, 2, 3, 4, 5, 4, 5 };
    float h_out[ROWS*COLS];

    float *d_values;
    int *d_csrOffsets;
    int *d_columns;
    float *d_out;

    hipMalloc((void **)&d_values, ROWS * BLOCK_SIZE * sizeof(float));
    hipMalloc((void **)&d_columns, ROWS * BLOCK_SIZE * sizeof(int));
    hipMalloc((void **)&d_csrOffsets, (ROWS + 1) * sizeof(int));
    hipMalloc((void **)&d_out, ROWS * COLS * sizeof(float));

    hipMemcpy(d_values, h_values, ROWS * BLOCK_SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_columns, h_columns, ROWS * BLOCK_SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_csrOffsets, h_csrOffsets, (ROWS + 1) * BLOCK_SIZE * sizeof(float), hipMemcpyHostToDevice);

    softmax<<<1, ROWS>>>(d_out, d_values, d_columns, d_csrOffsets, ROWS, COLS, BLOCK_SIZE);

    hipMemcpy(h_out, d_out, ROWS * COLS * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < ROWS; i++) {
        for (int j = 0; j < COLS; j++)
            printf("%f ", h_out[i * COLS + j]);
        printf("\n");
    }

    hipFree(d_values);
    hipFree(d_out);
    hipFree(d_csrOffsets);
    hipFree(d_columns);

    return 0;
}

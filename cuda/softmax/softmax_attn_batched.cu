
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define ROWS 6
#define COLS 6
#define BLOCK_SIZE 2
#define NUM_BATCH 2
//nvcc -o softmax_attn_batched softmax_attn_batched.cu

__global__ void softmax(float *d_out, float *d_values, int rows, int cols, int block_size, int num_batch)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= rows*num_batch)
        return;

    float sum = 0.0f;
    for (int k = 0; k < cols; k++){
        if (k < block_size){
            sum += expf(d_values[i * block_size + k]);
        }
        else{
            sum += expf(0);
        }
    }
    for (int j = 0; j < block_size; j++){
        d_out[i * block_size + j] = expf(d_values[i * block_size + j]) / sum;
    }
}

int main()
{
    //float h_values[ROWS*BLOCK_SIZE] = { 1.0f, 2.0f, 3.0f, 4.0f, 5.0f, 6.0f, 7.0f, 8.0f, 9.0f, 10.0f, 11.0f, 12.0f };
    float *h_values;
    //int   h_csrOffsets[]  = { 0, 2, 4, 6, 8, 10, 12 };
    //int   h_columns[]    = { 0, 1, 0, 1, 2, 3, 2, 3, 4, 5, 4, 5 };
    float h_out[ROWS*BLOCK_SIZE*NUM_BATCH];

    hipHostMalloc(&h_values, sizeof(float) * ROWS * BLOCK_SIZE * NUM_BATCH, hipHostMallocDefault);
    for(int j = 0; j < NUM_BATCH * ROWS * BLOCK_SIZE; j++) {
        h_values[j] = j+1; 
    }
    for (int k = 0; k < NUM_BATCH; k++) {
        for (int i = 0; i < ROWS; i++) {
            for (int j = 0; j < BLOCK_SIZE; j++)
                printf("%f ", h_values[i * BLOCK_SIZE + j + k * ROWS * BLOCK_SIZE]);
            printf("\n");
        }
        printf("\n");
    }

    float *d_values;
    float *d_out;

    hipMalloc((void **)&d_values, ROWS * BLOCK_SIZE * NUM_BATCH * sizeof(float));
    hipMalloc((void **)&d_out, ROWS * BLOCK_SIZE * NUM_BATCH * sizeof(float));

    hipMemcpy(d_values, h_values, ROWS * BLOCK_SIZE * NUM_BATCH * sizeof(float), hipMemcpyHostToDevice);

    softmax<<<NUM_BATCH, ROWS>>>(d_out, d_values, ROWS, COLS, BLOCK_SIZE, NUM_BATCH);

    hipMemcpy(h_out, d_out, ROWS * BLOCK_SIZE * NUM_BATCH * sizeof(float), hipMemcpyDeviceToHost);

    for (int k = 0; k < NUM_BATCH; k++) {
        for (int i = 0; i < ROWS; i++) {
            for (int j = 0; j < BLOCK_SIZE; j++)
                printf("%f ", h_out[i * BLOCK_SIZE + j + k * ROWS * BLOCK_SIZE]);
            printf("\n");
        }
        printf("\n");
    }

    hipFree(d_values);
    hipFree(d_out);

    return 0;
}

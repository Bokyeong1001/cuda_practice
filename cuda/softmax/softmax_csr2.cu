
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define ROWS 6
#define BLOCK_SIZE 2
//nvcc -o softmax_csr2 softmax_csr2.cu

__global__ void softmax(float *d_out, float *d_values, int *d_columns, int *d_csrOffsets, int rows, int block_size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= rows)
        return;

    float sum = 0.0f;
    for (int k = 0; k < block_size; k++){
        sum += expf(d_values[i * block_size + k]);

    }
    for (int j = 0; j < block_size; j++){
        d_out[i * block_size + j] = expf(d_values[i * block_size + j]) / sum;
    }
}

int main()
{
    float h_values[ROWS*BLOCK_SIZE] = { 10.0f, 2.0f, 13.0f, 4.0f, 1.0f, 6.0f, 8.0f, 8.0f, 5.0f, 10.0f, 31.0f, 22.0f };
    int   h_csrOffsets[]  = { 0, 2, 4, 6, 8, 10, 12 };
    int   h_columns[]    = { 0, 1, 0, 1, 2, 3, 2, 3, 4, 5, 4, 5 };
    float h_out[ROWS*BLOCK_SIZE];

    float *d_values;
    int *d_csrOffsets;
    int *d_columns;
    float *d_out;

    hipMalloc((void **)&d_values, ROWS * BLOCK_SIZE * sizeof(float));
    hipMalloc((void **)&d_columns, ROWS * BLOCK_SIZE * sizeof(int));
    hipMalloc((void **)&d_csrOffsets, (ROWS + 1) * sizeof(int));
    hipMalloc((void **)&d_out, ROWS * BLOCK_SIZE * sizeof(float));

    hipMemcpy(d_values, h_values, ROWS * BLOCK_SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_columns, h_columns, ROWS * BLOCK_SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_csrOffsets, h_csrOffsets, (ROWS + 1) * BLOCK_SIZE * sizeof(float), hipMemcpyHostToDevice);

    softmax<<<1, ROWS>>>(d_out, d_values, d_columns, d_csrOffsets, ROWS, BLOCK_SIZE);

    hipMemcpy(h_out, d_out, ROWS * BLOCK_SIZE * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < ROWS; i++) {
        for (int j = 0; j < BLOCK_SIZE; j++)
            printf("%f ", h_out[i * BLOCK_SIZE + j]);
        printf("\n");
    }

    hipFree(d_values);
    hipFree(d_out);
    hipFree(d_csrOffsets);
    hipFree(d_columns);

    return 0;
}

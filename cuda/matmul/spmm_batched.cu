
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h> // cudaMalloc, cudaMemcpy, etc.
#include <hipsparse.h>         // cusparseSpMM
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <math.h>             // fabs

//nvcc -lcusparse -o spmm_batched spmm_batched.cu

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

void print_matrix(const float *A, int nr_rows_A, int nr_cols_A, int batch_size) {
    for (int k = 0; k < batch_size; k++){
        for(int j = 0; j < nr_rows_A; j++){
            for(int i = 0; i < nr_cols_A; i++){
                int index = i*nr_rows_A + j + k*nr_rows_A*nr_cols_A;
                printf("%0.1f ",A[index]);
            }
            printf("\n");
        }
        printf("\n");
    }
}

void print_sparse_matrix(const float *values, const int *offsets, const int *columns, int rows, int cols, int nnz, int batch_size) {
    int k = 0;
    for(int b = 0; b < batch_size; b++){
        k=0;
        for(int i = 1; i < rows+1; i++){
            for(int j = 0; j < cols; j++){
                if(j==columns[k+nnz*b]){
                    if(k<offsets[i]){
                        printf("%0.1f ", values[k+nnz*b]);
                        k += 1;
                    }
                    else{
                        printf("0.0 ");
                    }
                }
                else{
                    printf("0.0 ");
                }
            }
            printf("\n");
        }
        printf("\n");
    }
    printf("\n");
}


int main(void) {
    // Host problem definition
    int   A_num_rows  = 4;
    int   A_num_cols  = 4;
    int   A_nnz       = 8;
    int   B_num_rows  = A_num_cols;
    int   B_num_cols  = 4;
    int   ldb         = B_num_rows;
    int   ldc         = A_num_rows;
    int   B_size      = ldb * B_num_cols;
    int   C_size      = ldc * B_num_cols;
    int   num_batches = 2;

    int   hA_csrOffsets[]  = { 0, 2, 4, 6, 8 };
    int   hA_columns[]    = { 0, 1, 0, 1, 2, 3, 2, 3,
                              0, 1, 0, 1, 2, 3, 2, 3 };
    float alpha            = 1.0f;
    float beta             = 0.0f;

    float *hA_values, *hB, *hC;
    hipHostMalloc(&hA_values, sizeof(float) * A_nnz  * num_batches, hipHostMallocDefault);
    hipHostMalloc(&hB, sizeof(float) * B_num_rows * B_num_cols * num_batches, hipHostMallocDefault);
    hipHostMalloc(&hC, sizeof(float) * A_num_rows * B_num_cols * num_batches, hipHostMallocDefault);

    for(int k=0; k<num_batches; k++) {
        for(int j=0; j<A_nnz; j++) {
            int index = k*A_nnz + j;
            hA_values[index] = index; 
        }
    }  

    for(int k=0; k<num_batches; k++) {
        for(int j=0; j<B_num_rows; j++) {
            for(int i=0; i<B_num_cols; i++) {
                int index = i*B_num_rows + j + k*B_num_rows*B_num_cols;
                hB[index] = i+j + 0.0f;
            }       
        }
    }
    printf("A = \n");
    print_sparse_matrix(hA_values, hA_csrOffsets, hA_columns, A_num_rows, A_num_cols, A_nnz, num_batches);

    printf("B = \n");
    print_matrix(hB, B_num_rows, B_num_cols,num_batches);
    //--------------------------------------------------------------------------
    // Device memory management
    int   *dA_csrOffsets, *dA_columns;
    float *dA_values, *dB, *dC;
    CHECK_CUDA( hipMalloc((void**) &dA_csrOffsets,
                           (A_num_rows + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dA_columns,
                           A_nnz * num_batches * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dA_values,
                           A_nnz * num_batches * sizeof(float)) )
    CHECK_CUDA( hipMalloc((void**) &dB,
                           B_size * num_batches * sizeof(float)) )
    CHECK_CUDA( hipMalloc((void**) &dC,
                           C_size * num_batches * sizeof(float)) )

    CHECK_CUDA( hipMemcpy(dA_csrOffsets, hA_csrOffsets,
                           (A_num_rows + 1) * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_columns, hA_columns, A_nnz * num_batches * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_values, hA_values, A_nnz * num_batches * sizeof(float),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dB, hB, B_size * num_batches * sizeof(float),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dC, hC, C_size * num_batches * sizeof(float),
                           hipMemcpyHostToDevice) )
    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB, matC;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create sparse matrix A in CSR format
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, A_num_rows, A_num_cols, A_nnz,
                                      dA_csrOffsets, dA_columns, dA_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    CHECK_CUSPARSE( hipsparseCsrSetStridedBatch(matA, num_batches, 0, A_nnz) )
    // Alternatively, the following code can be used for matA broadcast
    // CHECK_CUSPARSE( cusparseCsrSetStridedBatch(matA, num_batches, 0, 0) )
    // Create dense matrix B
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matB, A_num_cols, B_num_cols, ldb, dB,
                                        HIP_R_32F, HIPSPARSE_ORDER_COL) )
    CHECK_CUSPARSE( hipsparseDnMatSetStridedBatch(matB, num_batches, B_size) )
    // Create dense matrix C
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matC, A_num_rows, B_num_cols, ldc, dC,
                                        HIP_R_32F, HIPSPARSE_ORDER_COL) )
    CHECK_CUSPARSE( hipsparseDnMatSetStridedBatch(matC, num_batches, C_size) )

    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSpMM_bufferSize(
                                 handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SPMM_CSR_ALG2, &bufferSize) )
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

    // execute SpMM
    CHECK_CUSPARSE( hipsparseSpMM(handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SPMM_CSR_ALG2, dBuffer) )

    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matB) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matC) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    //--------------------------------------------------------------------------
    // device result check
    CHECK_CUDA( hipMemcpy(hC, dC, C_size * num_batches * sizeof(float),
                           hipMemcpyDeviceToHost) )
    printf("C = \n");
    print_matrix(hC, A_num_rows, B_num_cols,num_batches);
    //--------------------------------------------------------------------------
    // device memory deallocation
    CHECK_CUDA( hipFree(dBuffer) )
    CHECK_CUDA( hipFree(dA_csrOffsets) )
    CHECK_CUDA( hipFree(dA_columns) )
    CHECK_CUDA( hipFree(dA_values) )
    CHECK_CUDA( hipFree(dB) )
    CHECK_CUDA( hipFree(dC) )
    return EXIT_SUCCESS;
}
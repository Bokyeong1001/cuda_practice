#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipblas.h>
#include <hipblas.h>

using namespace std;
//nvcc -lcublas -o matmul_gemmEX_tensor matmul_gemmEX_tensor.cu

void blas_forward(const float *A, const float *B, float *C, const int m, const int n, const int k) {
    int lda=m,ldb=k,ldc=m;
    const float alf = 1;
    const float bet = 0;
    const float *alpha = &alf;
    const float *beta = &bet;
 
    // Create a handle for CUBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);
 
    // Do the actual multiplication
    hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, HIP_R_32F, lda, B, HIP_R_32F, ldb, beta, C, HIP_R_32F, ldc, HIPBLAS_COMPUTE_32F_FAST_16F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    
    // Destroy the handle
    hipblasDestroy(handle);
}


void print_matrix(float *A, int rows, int cols, int batch_size) {
    for (int i = 0; i < batch_size; i++){
        for (int j = 0; j < rows; j++){
            for(int k = 0; k < cols; k++){
                std::cout << A[i*rows*cols+k * rows + j] << " ";
            }
            std::cout << std::endl;
        }
        std::cout << std::endl;
    }

}

int main() {
     // Allocate 3 arrays on CPU
     int batch_size = 2; 
     int M = 2; 
     int K = 3;
     int N = 4;
 
 
     float *h_A, *h_B, *h_C;
     hipHostMalloc((void **) &h_A, sizeof(float) * M * K * batch_size);
     hipHostMalloc((void **) &h_B, sizeof(float) * K * N * batch_size);
     hipHostMalloc((void **) &h_C, sizeof(float) * M * N * batch_size);
     
     for (int i = 0; i < batch_size; i++){
        for (int j = 0; j < M * K; j++)
            h_A[i*M*K+j] = j;
        for (int j = 0; j < K * N; j++)
            h_B[i*K*N+j] = j + 4;
        for (int j = 0; j < M * N; j++)
            h_C[i*M*N+j] = 0;
     }
     
     // Allocate 3 arrays on GPU
     float *d_A, *d_B, *d_C;
     hipMalloc((void **) &d_A, sizeof(float)*M * K * batch_size);
     hipMalloc((void **) &d_B, sizeof(float)*K * N * batch_size);
     hipMalloc((void **) &d_C, sizeof(float)*M * N * batch_size);
 
     // copy matrix A and B from host to device memory
     hipMemcpy(d_A, h_A, sizeof(float)*M * K * batch_size, hipMemcpyHostToDevice);
     hipMemcpy(d_B, h_B, sizeof(float)*K * N * batch_size, hipMemcpyHostToDevice);

     std::cout << "A =" << std::endl;
     print_matrix(h_A, M, K, batch_size);

     std::cout << "B =" << std::endl;
     print_matrix(h_B, K, N, batch_size);
 
     // Multiply A and B on GPU
     blas_forward(d_A, d_B, d_C, M, N, K);
 
     // Copy (and print) the result on host memory
     hipMemcpy(h_C,d_C,M * N * sizeof(float) * batch_size,hipMemcpyDeviceToHost);
     std::cout << "C =" << std::endl;
     print_matrix(h_C, M, N, batch_size);

     //Free GPU memory
     hipFree(d_A);
     hipFree(d_B);
     hipFree(d_C);
 
     // Free CPU memory
     hipHostFree(h_A);
     hipHostFree(h_B);
     hipHostFree(h_C);
 
     return 0;
 }
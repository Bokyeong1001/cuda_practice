
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__
void multiply(int n, float * x, float * y, float * out)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) out[i] = x[i] * y[i];
}


void fill_array(float * data, int N){
    for(int idx=0;idx<N;idx++){
        data[idx] = idx;
    }
}

void print_output(float *x, float *y, float *out, int N){
    for(int idx=0;idx<N;idx++){
        printf("%.1f X %.1f = %.1f\n", x[idx], y[idx], out[idx]);
    }
}

int main(){
    int N = 10;

    float *h_a, *h_b, *h_c;
    hipHostMalloc((void **) &h_a, sizeof(float)*N, hipHostMallocDefault);
    hipHostMalloc((void **) &h_b, sizeof(float)*N, hipHostMallocDefault);
    hipHostMalloc((void **) &h_c, sizeof(float)*N, hipHostMallocDefault);

    fill_array(h_a,N);
    fill_array(h_b,N);
    
    float *d_a, *d_b, *d_c;
    hipMalloc((void **) &d_a, sizeof(float)*N);
    hipMalloc((void **) &d_b, sizeof(float)*N);
    hipMalloc((void **) &d_c, sizeof(float)*N);

    hipMemcpy(d_a, h_a, sizeof(float)*N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(float)*N, hipMemcpyHostToDevice);

    // Perform SAXPY on 1M elements
    multiply<<<1,N>>>(N, d_a, d_b, d_c);

    hipMemcpy(h_c, d_c, sizeof(float)*N, hipMemcpyDeviceToHost);
    print_output(h_a, h_b, h_c, N);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);
    return 0;
}
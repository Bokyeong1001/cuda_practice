
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h> // cudaMalloc, cudaMemcpy, etc.
#include <hipsparse.h>         // cusparseSpMM
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

//nvcc -lcusparse -o sddmm_batched sddmm_batched.cu
#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

void print_matrix(const float *A, int nr_rows_A, int nr_cols_A) {
 
    for(int i = 0; i < nr_rows_A; i++){
        for(int j = 0; j < nr_cols_A; j++){
            printf("%0.1f ",A[i * nr_cols_A + j]);
        }
        printf("\n");
    }
    printf("\n");
}

void print_matrixC(const float *C, const int *hC_offsets, const int *hC_columns, int nr_rows_C, int nr_cols_C) {
    int k = 0;
    for(int i = 1; i < nr_rows_C+1; i++){
        for(int j = 0; j < nr_cols_C; j++){
            if(j==hC_columns[k]){
                if(k<hC_offsets[i]){
                    printf("%0.1f ", C[k]);
                    k += 1;
                }
                else{
                    printf("0 ");
                }
            }
            else{
                printf("0 ");
            }
        }
        printf("\n");
    }
    printf("\n");
}

int main(void) {
    // Host problem definition
    int   A_num_rows   = 4;
    int   A_num_cols   = 4;
    int   B_num_rows   = A_num_cols;
    int   B_num_cols   = 3;
    int   C_nnz        = 9;
    int   lda          = A_num_cols;
    int   ldb          = B_num_cols;
    int   A_size       = lda * A_num_rows;
    int   B_size       = ldb * B_num_rows;
    int   num_batches  = 2;

    float hA1[]        = { 1.0f,   2.0f,  3.0f,  4.0f,
                           5.0f,   6.0f,  7.0f,  8.0f,
                           9.0f,  10.0f, 11.0f, 12.0f,
                           13.0f, 14.0f, 15.0f, 16.0f };
    float hA2[]        = { 10.0f,   11.0f,  12.0f,  13.0f,
                           14.0f,   15.0f,  16.0f,  17.0f,
                           18.0f,   19.0f,  20.0f,  21.0f,
                           22.0f,   23.0f,  24.0f,  25.0f };
    float hB1[]        = {  1.0f,  2.0f,  3.0f,
                            4.0f,  5.0f,  6.0f,
                            7.0f,  8.0f,  9.0f,
                            10.0f, 11.0f, 12.0f };
    float hB2[]        = {  6.0f,  4.0f,  2.0f,
                            3.0f,  7.0f,  1.0f,
                            9.0f,  5.0f,  2.0f,
                            8.0f,  4.0f,  7.0f };
    int   hC_offsets[]  = { 0, 3, 4, 7, 9 };
    int   hC_columns1[] = { 0, 1, 2, 1, 0, 1, 2, 0, 2 };
    int   hC_columns2[] = { 0, 1, 2, 0, 0, 1, 2, 1, 2 };
    float hC_values1[]  = { 0.0f, 0.0f, 0.0f, 0.0f, 0.0f,
                           0.0f, 0.0f, 0.0f, 0.0f };
    float hC_values2[]  = { 0.0f, 0.0f, 0.0f, 0.0f, 0.0f,
                           0.0f, 0.0f, 0.0f, 0.0f };
    float hC_result1[]  = { 70.0f, 80.0f, 90.0f,
                           184.0f,
                           246.0f, 288.0f, 330.0f,
                           334.0f, 450.0f };
    float hC_result2[]  = {305.0f, 229.0f, 146.0f,
                           409.0f,
                           513.0f, 389.0f, 242.0f,
                           469.0f, 290.0f };
    float alpha        = 1.0f;
    float beta         = 0.0f;
    //--------------------------------------------------------------------------
    // Device memory management
    int   *dC_offsets, *dC_columns;
    float *dC_values, *dB, *dA;
    CHECK_CUDA( hipMalloc((void**) &dA,
                           A_size * num_batches * sizeof(float)) )
    CHECK_CUDA( hipMalloc((void**) &dB,
                           B_size * num_batches * sizeof(float)) )
    CHECK_CUDA( hipMalloc((void**) &dC_offsets,
                           (A_num_rows + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dC_columns,
                           C_nnz * num_batches * sizeof(int))   )
    CHECK_CUDA( hipMalloc((void**) &dC_values,
                           C_nnz * num_batches * sizeof(float)) )

    CHECK_CUDA( hipMemcpy(dA, hA1, A_size * sizeof(float),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA + A_size, hA2, A_size * sizeof(float),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dB, hB1, B_size * sizeof(float),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dB + B_size, hB2, B_size * sizeof(float),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dC_offsets, hC_offsets,
                           (A_num_rows + 1) * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dC_columns, hC_columns1, C_nnz * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dC_columns + C_nnz, hC_columns2, C_nnz * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dC_values, hC_values1, C_nnz * sizeof(float),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dC_values + C_nnz, hC_values2, C_nnz * sizeof(float),
                           hipMemcpyHostToDevice) )
    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseDnMatDescr_t matA, matB;
    hipsparseSpMatDescr_t matC;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create dense matrix A
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matA, A_num_rows, A_num_cols, lda, dA,
                                        HIP_R_32F, HIPSPARSE_ORDER_ROW) )
    CHECK_CUSPARSE( hipsparseDnMatSetStridedBatch(matA, num_batches, A_size) )
    // Create dense matrix B
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matB, A_num_cols, B_num_cols, ldb, dB,
                                        HIP_R_32F, HIPSPARSE_ORDER_ROW) )
    CHECK_CUSPARSE( hipsparseDnMatSetStridedBatch(matB, num_batches, B_size) )
    // Create sparse matrix C in CSR format
    CHECK_CUSPARSE( hipsparseCreateCsr(&matC, A_num_rows, B_num_cols, C_nnz,
                                      dC_offsets, dC_columns, dC_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    CHECK_CUSPARSE( hipsparseCsrSetStridedBatch(matC, num_batches, 0, C_nnz) )
    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSDDMM_bufferSize(
                                 handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SDDMM_ALG_DEFAULT, &bufferSize) )
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

    // execute preprocess (optional)
    CHECK_CUSPARSE( hipsparseSDDMM_preprocess(
                                  handle,
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                  HIPSPARSE_SDDMM_ALG_DEFAULT, dBuffer) )
    // execute SpMM
    CHECK_CUSPARSE( hipsparseSDDMM(handle,
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                  HIPSPARSE_SDDMM_ALG_DEFAULT, dBuffer) )
    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matB) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matC) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    //--------------------------------------------------------------------------
    // device result check
    CHECK_CUDA( hipMemcpy(hC_values1, dC_values, C_nnz * sizeof(float),
                           hipMemcpyDeviceToHost) )
    CHECK_CUDA( hipMemcpy(hC_values2, dC_values + C_nnz, C_nnz * sizeof(float),
                           hipMemcpyDeviceToHost) )
    int correct = 1;
    for (int i = 0; i < C_nnz; i++) {
        if (hC_values1[i] != hC_result1[i]) {
            correct = 0; // direct floating point comparison is not reliable
            break;
        }
        if (hC_values2[i] != hC_result2[i]) {
            correct = 0; // direct floating point comparison is not reliable
            break;
        }
    }
    if (correct)
        printf("sddmm_csr_batched_example test PASSED\n");
    else
        printf("sddmm_csr_batched_example test FAILED: wrong result\n");
    //--------------------------------------------------------------------------
    // device memory deallocation
    CHECK_CUDA( hipFree(dBuffer) )
    CHECK_CUDA( hipFree(dA) )
    CHECK_CUDA( hipFree(dB) )
    CHECK_CUDA( hipFree(dC_offsets) )
    CHECK_CUDA( hipFree(dC_columns) )
    CHECK_CUDA( hipFree(dC_values) )
    return EXIT_SUCCESS;
}
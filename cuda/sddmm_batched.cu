
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h> // cudaMalloc, cudaMemcpy, etc.
#include <hipsparse.h>         // cusparseSpMM
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

//nvcc -lcusparse -o sddmm_batched sddmm_batched.cu
#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

void print_matrix(const float *A, int nr_rows_A, int nr_cols_A, int batch_size) {
    for (int k = 0; k < batch_size; k++){
        for(int j = 0; j < nr_rows_A; j++){
            for(int i = 0; i < nr_cols_A; i++){
                int index = j*nr_cols_A + i + k*nr_rows_A*nr_cols_A;
                printf("%0.1f ",A[index]);
            }
            printf("\n");
        }
        printf("\n");
    }
}

void print_matrixC(const float *C, const int *hC_offsets, const int *hC_columns, int nr_rows_C, int nr_cols_C, int C_nnz, int batch_size) {
    int k = 0;
    for(int b = 0; b < batch_size; b++){
        printf("\n%d\n", k);
        k=0;
        for(int i = 1; i < nr_rows_C+1; i++){
            for(int j = 0; j < nr_cols_C; j++){
                if(j==hC_columns[k+C_nnz*b]){
                    if(k<hC_offsets[i]){
                        printf("%0.1f ", C[k+C_nnz*b]);
                        k += 1;
                    }
                    else{
                        printf("0 ");
                    }
                }
                else{
                    printf("0 ");
                }
            }
            printf("\n");
        }
        printf("\n");
    }
    printf("\n");
}

int main(void) {
    // Host problem definition
    int   A_num_rows   = 4;
    int   A_num_cols   = 4;
    int   B_num_rows   = A_num_cols;
    int   B_num_cols   = 3;
    int   C_nnz        = 9;
    int   lda          = A_num_cols;
    int   ldb          = B_num_cols;
    int   A_size       = lda * A_num_rows;
    int   B_size       = ldb * B_num_rows;
    int   num_batches  = 2;

    int   hC_offsets[]  = { 0, 3, 4, 7, 9 };
    int   hC_columns[] = { 0, 1, 2, 1, 0, 1, 2, 0, 2, 
                           0, 1, 2, 1, 0, 1, 2, 0, 2};
    float alpha        = 1.0f;
    float beta         = 0.0f;

    float *hA, *hB, *hC_values;
    hipHostMalloc(&hA, sizeof(float) * A_num_rows * A_num_cols * num_batches, hipHostMallocDefault);
    hipHostMalloc(&hB, sizeof(float) * B_num_rows * B_num_cols * num_batches, hipHostMallocDefault);
    hipHostMalloc(&hC_values, sizeof(float) * C_nnz * num_batches, hipHostMallocDefault);

    for(int k=0; k<num_batches; k++) {
        for(int j=0; j<A_num_rows; j++) {
            for(int i=0; i<A_num_cols; i++) {
                int index = j*A_num_cols + i + k*A_num_rows*A_num_cols;
                hA[index] = i+j + 0.0f;
            }       
        }
    }  

    for(int k=0; k<num_batches; k++) {
        for(int j=0; j<B_num_rows; j++) {
            for(int i=0; i<B_num_cols; i++) {
                int index = j*B_num_cols + i + k*B_num_rows*B_num_cols;
                hB[index] = i+j + 0.0f;
            }       
        }
    }
    printf("A = \n");
    print_matrix(hA, A_num_rows, A_num_cols,num_batches);

    printf("B = \n");
    print_matrix(hB, B_num_rows, B_num_cols,num_batches);

    //--------------------------------------------------------------------------
    // Device memory management
    int   *dC_offsets, *dC_columns;
    float *dC_values, *dB, *dA;
    CHECK_CUDA( hipMalloc((void**) &dA,
                           A_size * num_batches * sizeof(float)) )
    CHECK_CUDA( hipMalloc((void**) &dB,
                           B_size * num_batches * sizeof(float)) )
    CHECK_CUDA( hipMalloc((void**) &dC_offsets,
                           (A_num_rows + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dC_columns,
                           C_nnz * num_batches * sizeof(int))   )
    CHECK_CUDA( hipMalloc((void**) &dC_values,
                           C_nnz * num_batches * sizeof(float)) )

    CHECK_CUDA( hipMemcpy(dA, hA, A_size * sizeof(float) * num_batches,
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dB, hB, B_size * sizeof(float) * num_batches,
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dC_offsets, hC_offsets,
                           (A_num_rows + 1) * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dC_columns, hC_columns, C_nnz * sizeof(int) * num_batches,
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dC_values, hC_values, C_nnz * sizeof(float) * num_batches,
                           hipMemcpyHostToDevice) )
    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseDnMatDescr_t matA, matB;
    hipsparseSpMatDescr_t matC;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create dense matrix A
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matA, A_num_rows, A_num_cols, lda, dA,
                                        HIP_R_32F, HIPSPARSE_ORDER_ROW) )
    CHECK_CUSPARSE( hipsparseDnMatSetStridedBatch(matA, num_batches, A_size) )
    // Create dense matrix B
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matB, A_num_cols, B_num_cols, ldb, dB,
                                        HIP_R_32F, HIPSPARSE_ORDER_ROW) )
    CHECK_CUSPARSE( hipsparseDnMatSetStridedBatch(matB, num_batches, B_size) )
    // Create sparse matrix C in CSR format
    CHECK_CUSPARSE( hipsparseCreateCsr(&matC, A_num_rows, B_num_cols, C_nnz,
                                      dC_offsets, dC_columns, dC_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    CHECK_CUSPARSE( hipsparseCsrSetStridedBatch(matC, num_batches, 0, C_nnz) )
    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSDDMM_bufferSize(
                                 handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SDDMM_ALG_DEFAULT, &bufferSize) )
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

    // execute preprocess (optional)
    CHECK_CUSPARSE( hipsparseSDDMM_preprocess(
                                  handle,
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                  HIPSPARSE_SDDMM_ALG_DEFAULT, dBuffer) )
    // execute SpMM
    CHECK_CUSPARSE( hipsparseSDDMM(handle,
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                  HIPSPARSE_SDDMM_ALG_DEFAULT, dBuffer) )
    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matB) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matC) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    //--------------------------------------------------------------------------
    // device result check
    CHECK_CUDA( hipMemcpy(hC_values, dC_values, C_nnz * sizeof(float) * num_batches,
                           hipMemcpyDeviceToHost) )
    printf("C = \n");
    /*for (int i = 0; i < C_nnz; i++) {
        printf("%0.1f ", hC_values[i]);
    }*/
    print_matrixC(hC_values, hC_offsets, hC_columns, A_num_rows, B_num_cols, C_nnz, num_batches);

    //--------------------------------------------------------------------------
    // device memory deallocation
    CHECK_CUDA( hipFree(dBuffer) )
    CHECK_CUDA( hipFree(dA) )
    CHECK_CUDA( hipFree(dB) )
    CHECK_CUDA( hipFree(dC_offsets) )
    CHECK_CUDA( hipFree(dC_columns) )
    CHECK_CUDA( hipFree(dC_values) )
    return EXIT_SUCCESS;
}
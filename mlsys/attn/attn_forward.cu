
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h> // cudaMalloc, cudaMemcpy, etc.
#include <hipsparse.h>         // cusparseSpMM
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

//nvcc -Xcompiler -fPIC -shared -lcusparse -o attn_forward.so attn_forward.cu

__global__ void softmax_kernel(float *d_out, float *d_values, int rows, int cols, int block_size, int num_batch)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= rows*num_batch)
        return;

    float max = 0.0;
    for (int k = 0; k < block_size; k++){
        if (max < d_values[i * block_size + k]){
            max = d_values[i * block_size + k];
        }
    }
    float sum = 0.0f;
    for (int k = 0; k < cols; k++){
        if (k < block_size){
            sum += expf(d_values[i * block_size + k] - max);
        }
        else{
            sum += expf(0 - max);
        }
    }
    for (int k = 0; k < block_size; k++){
        d_out[i * block_size + k] = expf(d_values[i * block_size + k] - max) / sum;
    }
}

void spmm(hipsparseHandle_t handle, void *dBuffer, float *dAttn, float *dValue, float *dOut, int *d_offsets, int *d_columns, int seq_len, int emb_dim, int nnz, int num_batches)
{
    // Host problem definition
    int   ldb         = seq_len;
    int   ldc         = seq_len;
    int   output_size = seq_len * emb_dim;

    float alpha            = 1.0f;
    float beta             = 0.0f;

    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB, matC;
    size_t               bufferSize = 0;

    // Create sparse matrix A in CSR format
    hipsparseCreateCsr(&matA, seq_len, seq_len, nnz,
                                      d_offsets, d_columns, dAttn,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
    hipsparseCsrSetStridedBatch(matA, num_batches, 0, nnz);
    // Alternatively, the following code can be used for matA broadcast
    // cusparseCsrSetStridedBatch(matA, num_batches, 0, 0);
    // Create dense matrix B
    hipsparseCreateDnMat(&matB, seq_len, emb_dim, ldb, dValue,
                                        HIP_R_32F, HIPSPARSE_ORDER_COL);
    hipsparseDnMatSetStridedBatch(matB, num_batches, output_size);
    // Create dense matrix C
    hipsparseCreateDnMat(&matC, seq_len, emb_dim, ldc, dOut,
                                        HIP_R_32F, HIPSPARSE_ORDER_COL);
    hipsparseDnMatSetStridedBatch(matC, num_batches, output_size);

    // allocate an external buffer if needed
    hipsparseSpMM_bufferSize(
                                 handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SPMM_CSR_ALG2, &bufferSize);
    hipMalloc(&dBuffer, bufferSize);

    // execute SpMM
    hipsparseSpMM(handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SPMM_CSR_ALG2, dBuffer);

    // destroy matrix/vector descriptors
    hipsparseDestroySpMat(matA);
    hipsparseDestroyDnMat(matB);
    hipsparseDestroyDnMat(matC);
    hipFree(dBuffer);
}

void sddmm(hipsparseHandle_t handle, void *dBuffer, float *dQuery, float *dKey, float *dAttn, int *d_offsets, int *d_columns, int seq_len, int emb_dim, int nnz, int num_batches){
    
    size_t bufferSize  = 0;
    int   lda          = emb_dim;
    int   ldb          = seq_len;
    int   input_size       = seq_len * emb_dim;

    float alpha            = 1.0f;
    float beta             = 0.0f;

    hipsparseDnMatDescr_t matA, matB;
    hipsparseSpMatDescr_t matC;
    
    // Create dense matrix A
    hipsparseCreateDnMat(&matA, seq_len, emb_dim, lda, dQuery,
                                        HIP_R_32F, HIPSPARSE_ORDER_ROW);
    hipsparseDnMatSetStridedBatch(matA, num_batches, input_size);
    // Create dense matrix B
    hipsparseCreateDnMat(&matB, emb_dim, seq_len, ldb, dKey,
                                        HIP_R_32F, HIPSPARSE_ORDER_ROW);
    hipsparseDnMatSetStridedBatch(matB, num_batches, input_size);
    // Create sparse matrix C in CSR format
    hipsparseCreateCsr(&matC, seq_len, seq_len, nnz,
                                      d_offsets, d_columns, dAttn,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
    hipsparseCsrSetStridedBatch(matC, num_batches, 0, nnz);
    // allocate an external buffer if needed
    hipsparseSDDMM_bufferSize(
                                 handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SDDMM_ALG_DEFAULT, &bufferSize);
    hipMalloc(&dBuffer, bufferSize);

    // execute preprocess (optional)
    hipsparseSDDMM_preprocess(
                                  handle,
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                  HIPSPARSE_SDDMM_ALG_DEFAULT, dBuffer);
    // execute SpMM
    hipsparseSDDMM(handle,
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                  HIPSPARSE_SDDMM_ALG_DEFAULT, dBuffer);
    // destroy matrix/vector descriptors
    hipsparseDestroyDnMat(matA);
    hipsparseDestroyDnMat(matB);
    hipsparseDestroySpMat(matC);
    hipFree(dBuffer);
}

extern "C" {
void attn_forward(float *hQuery, float *hKey, float *hValue, float *hAttn, float *hAttnOut, float *hOut, int *hOffsets, int *hColumns, int seq_len, int emb_dim, int nnz, int block_size, int num_batches)
{
    // Host problem definition
    int   input_size       = seq_len * emb_dim;

    int   *d_offsets, *d_columns;
    float *dQuery, *dKey, *dValue, *dAttn, *dAttnOut, *dOut;

    hipMalloc((void**) &dQuery, input_size * num_batches * sizeof(float));
    hipMalloc((void**) &dKey, input_size * num_batches * sizeof(float));
    hipMalloc((void**) &dValue, input_size * num_batches * sizeof(float));
    hipMalloc((void**) &dAttn, nnz * num_batches * sizeof(float));
    hipMalloc((void**) &dAttnOut, nnz * num_batches * sizeof(float));
    hipMalloc((void**) &d_offsets, (seq_len + 1) * sizeof(int));
    hipMalloc((void**) &d_columns, nnz * num_batches * sizeof(int));
    hipMalloc((void**) &dOut, input_size * num_batches * sizeof(float));

    hipMemcpy(dQuery, hQuery, input_size * sizeof(float) * num_batches, hipMemcpyHostToDevice);
    hipMemcpy(dKey, hKey, input_size * sizeof(float) * num_batches, hipMemcpyHostToDevice);
    hipMemcpy(dValue, hValue, input_size * sizeof(float) * num_batches, hipMemcpyHostToDevice);
    hipMemcpy(dAttn, hAttn, nnz * sizeof(float) * num_batches, hipMemcpyHostToDevice);
    hipMemcpy(dAttnOut, hAttnOut, nnz * sizeof(float) * num_batches, hipMemcpyHostToDevice);
    hipMemcpy(d_offsets, hOffsets, (seq_len + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_columns, hColumns, nnz * sizeof(int) * num_batches, hipMemcpyHostToDevice);
    hipMemcpy(dOut, hOut, input_size * sizeof(float) * num_batches, hipMemcpyHostToDevice);

    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseCreate(&handle);
    void* dBuffer    = NULL;

    sddmm(handle, dBuffer, dQuery, dKey, dAttn, d_offsets, d_columns, seq_len, emb_dim, nnz, num_batches);
    hipMemcpy(hAttn, dAttn, nnz * sizeof(float) * num_batches, hipMemcpyDeviceToHost);

    softmax_kernel<<<num_batches, seq_len>>>(dAttnOut, dAttn, seq_len, seq_len, block_size, num_batches);
    hipMemcpy(hAttnOut, dAttnOut, nnz * sizeof(float) * num_batches, hipMemcpyDeviceToHost);

    spmm(handle, dBuffer, dAttnOut, dValue, dOut, d_offsets, d_columns, seq_len, emb_dim, nnz, num_batches);
    hipMemcpy(hOut, dOut, input_size * sizeof(float) * num_batches, hipMemcpyDeviceToHost);
    //--------------------------------------------------------------------------
    // device memory deallocation
    
    hipFree(dQuery);
    hipFree(dKey);
    hipFree(dValue);
    hipFree(dAttn);
    hipFree(dAttnOut);
    hipFree(dOut);
    hipFree(d_offsets);
    hipFree(d_columns);
    
    hipsparseDestroy(handle);
}
}

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define BLOCK_SIZE 4
// nvcc -Xcompiler -fPIC -shared -o cuda_matmul.so cuda_matmul.cu

 __global__ void gpu_matrix_mult(float *A, float *B, float *C, int N)
 { 
    int ROW = blockIdx.y*blockDim.y+threadIdx.y;
    int COL = blockIdx.x*blockDim.x+threadIdx.x;

    float tmpSum = 0;

    if (ROW < N && COL < N) {
        // each thread computes one element of the block sub-matrix
        for (int i = 0; i < N; i++) {
            tmpSum += A[ROW * N + i] * B[i * N + COL];
        }
    }
    C[ROW * N + COL] = tmpSum;
 } 

extern "C" {
void cuda_matmul(float *a, float *b, float *c, int N, size_t size)
{

    float *d_a, *d_b, *d_c;

    hipMalloc((void **)&d_a, size * sizeof(float));
    hipMalloc((void **)&d_b, size * sizeof(float));
    hipMalloc((void **)&d_c, size * sizeof(float));

    hipMemcpy(d_a, a, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * sizeof(float), hipMemcpyHostToDevice);

    unsigned int grid_rows = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    gpu_matrix_mult<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, N);    

    hipMemcpy(c, d_c, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
}
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipblas.h>
#include <hipblas.h>

#define BLOCK_SIZE 4
//nvcc -Xcompiler -fPIC -shared -lcublas -o cuda_matmul_sgemm.so cuda_matmul_sgemm.cu

void gpu_blas_mmul(const float *A, const float *B, float *C, const int m, const int n, const int k) {
    int lda=m,ldb=k,ldc=m;
    const float alf = 1;
    const float bet = 0;
    const float *alpha = &alf;
    const float *beta = &bet;
 
    // Create a handle for CUBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);
 
    // Do the actual multiplication
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
 
    // Destroy the handle
    hipblasDestroy(handle);
}

extern "C" {
void cuda_matmul(float *a, float *b, float *c, int m, int n, int k, size_t a_size, size_t b_size, size_t c_size)
{
    float *d_a, *d_b, *d_c;

    hipMalloc((void **)&d_a, a_size * sizeof(float));
    hipMalloc((void **)&d_b, b_size * sizeof(float));
    hipMalloc((void **)&d_c, c_size * sizeof(float));

    hipMemcpy(d_a, a, a_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, b_size * sizeof(float), hipMemcpyHostToDevice);

    gpu_blas_mmul(d_a, d_b, d_c, m, n, k);

    hipMemcpy(c, d_c, c_size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
}